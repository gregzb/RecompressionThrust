#include "hip/hip_runtime.h"
#include <thrust/adjacent_difference.h>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/device_vector.h>
#include <thrust/gather.h>
#include <thrust/generate.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/transform_output_iterator.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/transform_scan.h>
#include <thrust/unique.h>
#include <thrust/zip_function.h>

#include "arena.cuh"
#include "types.hpp"
#include "util.hpp"
#include <chrono>
#include <iostream>

#include "mio/mio.hpp"

namespace Cu
{
    void init()
    {
        hipSetDevice(0);
        hipFree(0);
        hipDeviceSynchronize();
    }
    template <bool IS_GPU>
    struct Thrust
    {
        struct efficient_random_byte_functor
        {
            unsigned int seed;

            __host__ __device__ efficient_random_byte_functor(unsigned int _seed) : seed(_seed) {}

            // A simple 32-bit hash function based on xorshift and multiplicative
            // mixing.
            __host__ __device__ unsigned int hash(unsigned int x) const
            {
                x ^= x >> 16;
                x *= 0x85ebca6b;
                x ^= x >> 13;
                x *= 0xc2b2ae35;
                x ^= x >> 16;
                return x;
            }

            // For each index, mix the seed and index to produce a pseudo-random number.
            __host__ __device__ unsigned char operator()(unsigned int idx) const
            {
                unsigned int key = idx * 0x9E3779B9;
                unsigned int h = hash(seed ^ key);
                return static_cast<unsigned char>((h % 17) & 0b1);
            }
        };

        // // using InputIterT =
        // SizedIter<decltype(thrust::make_permutation_iterator(static_cast<symbol_t
        // *>(nullptr),
        // thrust::make_transform_iterator(thrust::make_counting_iterator(0),
        // stride_functor(0))))>;
        // // using RulesIterT =
        // SizedIter<decltype(thrust::make_permutation_iterator(static_cast<thrust::tuple<symbol_t,
        // symbol_t> *>(nullptr),
        // thrust::make_transform_iterator(thrust::make_counting_iterator(0),
        // stride_functor(0))))>; using InputIterT =
        // SizedIter<decltype(thrust::make_permutation_iterator(thrust::device_pointer_cast<symbol_t>(nullptr),
        // thrust::make_transform_iterator(thrust::make_counting_iterator(0),
        // stride_functor(0))))>; using RulesIterT =
        // SizedIter<decltype(thrust::make_permutation_iterator(thrust::device_pointer_cast<thrust::tuple<symbol_t,
        // symbol_t>>(nullptr),
        // thrust::make_transform_iterator(thrust::make_counting_iterator(0),
        // stride_functor(0))))>;

        using InputIterType = typename std::conditional<
            IS_GPU,
            decltype(thrust::make_permutation_iterator(
                thrust::device_pointer_cast<symbol_t>(nullptr),
                thrust::make_transform_iterator(thrust::make_counting_iterator(0), stride_functor(0)))),
            decltype(thrust::make_permutation_iterator(
                static_cast<symbol_t *>(nullptr),
                thrust::make_transform_iterator(thrust::make_counting_iterator(0), stride_functor(0))))>::type;

        // Select the correct Rules iterator type using std::conditional.
        using RulesIterType = typename std::conditional<
            IS_GPU,
            decltype(thrust::make_permutation_iterator(
                thrust::device_pointer_cast<thrust::tuple<symbol_t, symbol_t>>(nullptr),
                thrust::make_transform_iterator(thrust::make_counting_iterator(0), stride_functor(0)))),
            decltype(thrust::make_permutation_iterator(
                static_cast<thrust::tuple<symbol_t, symbol_t> *>(nullptr),
                thrust::make_transform_iterator(thrust::make_counting_iterator(0), stride_functor(0))))>::type;

        // Now wrap the types in SizedIter.
        using InputIterT = SizedIter<InputIterType>;
        using RulesIterT = SizedIter<RulesIterType>;

        enum class CurrentInputPos
        {
            TWO,
            THREE
        };

        static CurrentInputPos other_pos(CurrentInputPos curr_input_pos)
        {
            return curr_input_pos == CurrentInputPos::TWO ? CurrentInputPos::THREE : CurrentInputPos::TWO;
        }

        // template <bool IS_GPU>
        static InputIterT view_for(Arena<IS_GPU> &arena, CurrentInputPos curr_input_pos, symbol_t original_size,
                                   symbol_t curr_input_size)
        {
            symbol_t mult = curr_input_pos == CurrentInputPos::TWO ? 2 : 3;
            return arena.template view_start_at_items<symbol_t>(mult * original_size, curr_input_size);
        }

        struct make_key_count_tuple
        {
            __host__ __device__ thrust::tuple<uint64_t, symbol_t> operator()(symbol_t key, symbol_t count,
                                                                             symbol_t index) const
            {
                return thrust::make_tuple((((uint64_t)(uint32_t)key) << 32) | count, index);
            }
        };

        struct count_predicate
        {
            __host__ __device__ bool operator()(symbol_t key, symbol_t count, symbol_t index) const { return count > 1; }
        };

        static symbol_t bcomp(Arena<IS_GPU> &arena, InputIterT &curr_input, CurrentInputPos &curr_input_pos,
                              symbol_t original_size, symbol_t num_symbols, RulesIterT rules)
        {
            using std::chrono::duration;
            using std::chrono::duration_cast;
            using std::chrono::high_resolution_clock;
            using std::chrono::nanoseconds;

            auto keys = view_for(arena, other_pos(curr_input_pos), original_size, curr_input.size);
            auto counts = arena.template view_start_at_items<symbol_t>(4 * original_size, curr_input.size);
            auto indices = view_for(arena, curr_input_pos, original_size, curr_input.size);
            auto keys_counts_double =
                arena.template view_start_at_bytes<uint64_t>(0 * original_size * sizeof(symbol_t), curr_input.size);

            auto [keys_end, counts_end] = thrust::reduce_by_key(curr_input.iter, curr_input.iter + curr_input.size,
                                                                thrust::make_constant_iterator(1), keys.iter, counts.iter);

            keys.shrink(keys_end - keys.iter);

            if (keys.size == curr_input.size)
            {
                return num_symbols;
            }

            auto zip_iter = thrust::make_zip_iterator(keys.iter, counts.iter, thrust::make_counting_iterator(0));
            auto keys_counts_and_indices = thrust::make_zip_iterator(keys_counts_double.iter, indices.iter);
            auto transform_it = thrust::make_transform_output_iterator(keys_counts_and_indices,
                                                                       thrust::make_zip_function(make_key_count_tuple()));
            auto transform_it_end =
                thrust::copy_if(zip_iter, zip_iter + keys.size, transform_it, thrust::make_zip_function(count_predicate()));

            symbol_t num_blocks = transform_it_end - transform_it;

            if (num_blocks > 0)
            {
                thrust::sort_by_key(keys_counts_double.iter, keys_counts_double.iter + num_blocks, indices.iter);

                auto offsets = arena.template view_start_at_items<symbol_t>(4 * original_size, num_blocks);

                offsets.iter[0] = 0;
                auto prev_and_next = thrust::make_zip_iterator(keys_counts_and_indices, keys_counts_and_indices + 1);

                thrust::transform_inclusive_scan(
                    prev_and_next, prev_and_next + offsets.size - 1, offsets.iter + 1,
                    [] __host__ __device__(
                        const thrust::tuple<thrust::tuple<uint64_t, symbol_t>, thrust::tuple<uint64_t, symbol_t>>
                            &item) -> symbol_t
                    {
                        bool eq = thrust::get<0>(thrust::get<0>(item)) == thrust::get<0>(thrust::get<1>(item));
                        return (symbol_t)(!eq);
                    },
                    [] __host__ __device__(symbol_t a, symbol_t b) -> symbol_t
                    { return a + b; });

                thrust::unique_by_key_copy(
                    offsets.iter, offsets.iter + offsets.size,
                    thrust::make_transform_iterator(
                        keys_counts_double.iter,
                        [] __host__ __device__(uint64_t item) -> thrust::tuple<symbol_t, symbol_t>
                        {
                            return thrust::make_tuple(item >> 32, item & 0xffffffff);
                        }),
                    thrust::make_discard_iterator(), rules.iter + num_symbols);

                symbol_t last_offset = offsets.iter[offsets.size - 1];

                auto offset_iter = thrust::make_transform_iterator(
                    offsets.iter,
                    [num_symbols] __host__ __device__(const symbol_t &offset) -> symbol_t
                    { return num_symbols + offset; });
                symbol_t num_new_symbols = last_offset + 1;
                num_symbols += num_new_symbols;

                thrust::scatter(offset_iter, offset_iter + offsets.size, indices.iter, keys.iter);
            }
            curr_input = keys;
            curr_input_pos = other_pos(curr_input_pos);
            return num_symbols;
        }

        inline static size_t total_pcomp_sorted = 0;

        static symbol_t pcomp(Arena<IS_GPU> &arena, InputIterT &curr_input, CurrentInputPos curr_input_pos,
                              symbol_t original_size, symbol_t num_symbols, RulesIterT rules, int x)
        {
            auto pseudo_rand_bits = arena.template view_start_at_bytes<uint8_t>(
                original_size / 2 * sizeof(symbol_t) * 3 + 0 * original_size, num_symbols);
            auto counting_iter = thrust::make_counting_iterator(0);

            thrust::transform(counting_iter, counting_iter + pseudo_rand_bits.size, pseudo_rand_bits.iter,
                              efficient_random_byte_functor(num_symbols + x));

            auto assigned_bit = arena.template view_start_at_bytes<uint8_t>(
                original_size / 2 * sizeof(symbol_t) * 3 + 1 * original_size, curr_input.size);

            thrust::gather(curr_input.iter, curr_input.iter + curr_input.size, pseudo_rand_bits.iter,
                           assigned_bit.iter);

            thrust::adjacent_difference(
                assigned_bit.iter, assigned_bit.iter + assigned_bit.size, assigned_bit.iter,
                [] __host__ __device__(uint8_t curr, uint8_t prev) -> uint8_t
                { return curr && !prev; });

            assigned_bit.iter[0] = 0;

            auto pairs = arena.template view_start_at_bytes<uint64_t>(0, assigned_bit.size / 2);
            auto indices = arena.template view_start_at_bytes<symbol_t>(sizeof(uint64_t) * original_size / 2 * 1,
                                                                        assigned_bit.size / 2);

            auto iter_adj =
                thrust::make_zip_iterator(curr_input.iter, curr_input.iter + 1, thrust::make_counting_iterator(1));

            auto pairs_and_sources_iter = thrust::make_zip_iterator(pairs.iter, indices.iter);
            auto pairs_and_sources_transformed_iter = thrust::make_transform_output_iterator(
                pairs_and_sources_iter, thrust::make_zip_function(make_key_count_tuple()));
            {
                auto pairs_and_sources_transformed_end = thrust::copy_if(iter_adj, iter_adj + assigned_bit.size - 1, assigned_bit.iter + 1,
                                                                         pairs_and_sources_transformed_iter, thrust::identity<uint8_t>());

                pairs.shrink(pairs_and_sources_transformed_end - pairs_and_sources_transformed_iter);
                indices.shrink(pairs_and_sources_transformed_end - pairs_and_sources_transformed_iter);
            }

            if (pairs.size > 0)
            {

                thrust::sort_by_key(pairs.iter, pairs.iter + pairs.size, indices.iter);

                auto offsets = arena.template view_start_at_items<symbol_t>(4 * original_size, pairs.size);

                auto prev_and_next = thrust::make_zip_iterator(pairs.iter, pairs.iter + 1);
                thrust::transform_inclusive_scan(
                    prev_and_next, prev_and_next + pairs.size - 1, offsets.iter + 1,
                    [] __host__ __device__(const thrust::tuple<uint64_t, uint64_t> &item) -> symbol_t
                    {
                        bool eq = thrust::get<0>(item) == thrust::get<1>(item);
                        return (symbol_t)(!eq);
                    },
                    thrust::plus<symbol_t>());

                thrust::unique_by_key_copy(offsets.iter, offsets.iter + offsets.size,
                                           thrust::make_transform_iterator(
                                               pairs.iter,
                                               [] __host__ __device__(uint64_t item) -> thrust::tuple<symbol_t, symbol_t>
                                               {
                                                   return thrust::make_tuple(item >> 32, item & 0xffffffff);
                                               }),
                                           thrust::make_discard_iterator(), rules.iter + num_symbols);

                auto offset_iter = thrust::make_transform_iterator(
                    offsets.iter,
                    [num_symbols] __host__ __device__(const symbol_t &offset) -> symbol_t
                    { return num_symbols + offset; });
                symbol_t num_new_symbols = offsets.iter[offsets.size - 1] + 1;
                num_symbols += num_new_symbols;

                thrust::scatter(offset_iter, offset_iter + offsets.size, indices.iter, curr_input.iter);

                auto curr_input_and_idx = thrust::make_zip_iterator(curr_input.iter, thrust::make_counting_iterator(0));
                if constexpr (IS_GPU)
                {
                    auto assigned_bit_raw_ptr = thrust::device_pointer_cast(&assigned_bit.iter[0]);
                    auto new_end = thrust::remove_if(
                        curr_input_and_idx, curr_input_and_idx + curr_input.size,
                        [assigned_bit_raw_ptr] __host__ __device__(const thrust::tuple<symbol_t, symbol_t> &item)
                        {
                            auto idx = thrust::get<1>(item);
                            return *(assigned_bit_raw_ptr + idx + 1) == 1;
                        });

                    curr_input.shrink(new_end - curr_input_and_idx);
                }
                else
                {
                    auto assigned_bit_raw_ptr = &assigned_bit.iter[0];
                    auto new_end = thrust::remove_if(
                        curr_input_and_idx, curr_input_and_idx + curr_input.size,
                        [assigned_bit_raw_ptr] __host__ __device__(const thrust::tuple<symbol_t, symbol_t> &item)
                        {
                            auto idx = thrust::get<1>(item);
                            return *(assigned_bit_raw_ptr + idx + 1) == 1;
                        });

                    curr_input.shrink(new_end - curr_input_and_idx);
                }
            }
            return num_symbols;
        }

        using iter_t = const char *;
        static void recompression(symbol_t alphabet_size, iter_t begin, iter_t end)
        {
            using std::chrono::duration;
            using std::chrono::duration_cast;
            using std::chrono::high_resolution_clock;
            using std::chrono::nanoseconds;
            symbol_t num_symbols = alphabet_size;

            symbol_t input_size = end - begin;

            symbol_t unit_size = std::max(alphabet_size * 4, input_size);

            Arena<IS_GPU> arena(unit_size * sizeof(symbol_t) * 7);
            auto initial_input = arena.template view_start_at_items<char>(0 * unit_size, input_size);
            auto curr_input = arena.template view_start_at_items<symbol_t>(2 * unit_size, input_size);

            auto curr_input_pos = CurrentInputPos::TWO;

            auto rules = arena.template view_start_at_bytes<thrust::tuple<symbol_t, symbol_t>>(
                5 * unit_size * sizeof(symbol_t), input_size);

            time_f_print_void([&]()
                              {
                thrust::copy(begin, end, initial_input.iter);
                thrust::copy(initial_input.iter, initial_input.iter + initial_input.size, curr_input.iter); },
                              "read file and transfer to a convenient spot in memory");

            int num_layers = 0;
            while (curr_input.size > 1)
            {
                num_symbols = bcomp(arena, curr_input, curr_input_pos, unit_size, num_symbols, rules);

                num_layers++;

                if (curr_input.size == 1)
                    break;

                int cnt = 0;
                while (true)
                {
                    size_t prev_size = curr_input.size;
                    auto new_num_symbols2 = pcomp(arena, curr_input, curr_input_pos, unit_size, num_symbols, rules, cnt);
                    if (curr_input.size != prev_size)
                    {
                        num_symbols = new_num_symbols2;
                        break;
                    }
                    cnt++;
                }
                num_layers++;
            }

            rules.shrink(num_symbols);
        }
    };
    template struct Thrust<false>;
    template struct Thrust<true>;
} // namespace Cu